#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib> 
#include <cstring> 
#include "timer.h"
#include <iostream>


#define H 1024
#define W 1024
#define C 3 //input channels
#define FW 3
#define FH 3
#define K 64 //output channels
#define P 1 //padding 
double *d_I,*d_F,*d_O, *h_I,*h_F,*h_O;
double *d_Io,*h_Io;

__global__ void convolution(double *I,double *F, double *O){

}


int main(int argc, char* argv[]){
    
    size_t size_I = H*W*C;
    size_t size_Io =  (H+2*P)*(W+2*P)*C;
    size_t size_F = FH*FW*C*K;
    size_t size_O = K*H*W;
    printf("Malloc \n");
    h_I = (double*)malloc(size_I*sizeof(double));
    h_F = (double*)malloc(size_F*sizeof(double));
    h_O = (double*)malloc(size_O*sizeof(double));
    h_Io = (double*)malloc(size_Io*sizeof(double));
    printf("init?\n");
    // init I tensor
    // Initialize I
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < W; ++x) {
            for (int y = 0; y < H; ++y) {
                h_I[c * W * H + x * H + y] = c * (x + y);
            }
        }
    }
    printf("init->F\n");
    // Initialize F filter
    for (int k = 0; k < K; ++k) {
        for (int c = 0; c < C; ++c) {
            for (int i = 0; i < FH; ++i) {
                for (int j = 0; j < FW; ++j) {
                    h_F[k * C * FH * FW + c * FH * FW + i * FW + j] = (c + k) * (i + j);
                }
            }
        }
    }
    printf("init->Io\n");
    // Initialize I0 with padding
    for (int c = 0; c < C; ++c) {
        for (int x = 0; x < W + 2 * P; ++x) {
            for (int y = 0; y < H + 2 * P; ++y) {
                if (x == 0 || y == 0 || x == W + 2 * P - 1 || y == H + 2 * P - 1) {
                    h_Io[c * (W + 2 * P) * (H + 2 * P) + x * (H + 2 * P) + y] = 0;
                } else {
                    h_Io[c * (W + 2 * P) * (H + 2 * P) + x * (H + 2 * P) + y] = h_I[c * W * H + (x - 1) * H + (y - 1)];
                }
            }
        }
    }
   printf("cuddda \n");
    hipMalloc(&d_I,size_I*sizeof(double));
    hipMalloc(&d_F,size_F*sizeof(double));
    hipMalloc(&d_O,size_O*sizeof(double));
    hipMalloc(&d_Io,size_Io*sizeof(double));

    hipMemcpy(d_Io,h_Io,size_Io * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_F,h_F,size_F * sizeof(double),hipMemcpyHostToDevice);

    dim3 dimBlock(H); //1024
    dim3 dimGrid(K,H); //64,1024
    // warm-up
    printf("HIIII \n");
    convolution<<<dimGrid, dimBlock>>>(d_Io, d_F, d_O);
    hipDeviceSynchronize();
    //
    printf("warmupppp done");
    initialize_timer();
    start_timer();
    convolution<<<dimGrid, dimBlock>>>(d_Io, d_F, d_O);
    hipDeviceSynchronize();
    stop_timer();
    double time = elapsed_time();
    printf( "Time: %lf (sec)\n",time);

    //printf( "Time: %lf (sec), nFlops: %0.0lf, GFlopsS: %lf\n",
    //time, nFlops, nGFlopsPerSec);
    hipFree(d_I);
    hipFree(d_Io);
    hipFree(d_F);
    hipFree(d_O);
    return 0;
}