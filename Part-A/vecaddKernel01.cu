#include "hip/hip_runtime.h"
/*
/// vecAddKernel00.cu
/// For COMS E6998 Spring 2024
/// Instructor: Kaoutar El Maghraoui
/// By Yonathan Daniel 
Created: 2024-03-14
/// This Kernel adds two Vectors A and B in C on GPU
/// with coalesced memory access.
/// 
*/
#include <stdio.h>
__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    int stide = blockDim.x*gridDim.x;
    for(int i = index; i< N; i+=stride )
    {
        C[i] = A[i]+B[i]
    }
}