#include "hip/hip_runtime.h"
/*
For COMS E6998(HPML) Spring 2024
Modified by Yonathan Daniel
*/
#include "matmultKernel.h"

#define FOOTPRINT_SIZE BLOCK_SIZE

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

}