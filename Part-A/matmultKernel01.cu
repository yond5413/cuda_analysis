#include "hip/hip_runtime.h"
/*
For COMS E6998(HPML) Spring 2024
Modified by Yonathan Daniel
*/
#include <stdio.h> // for debugging

#include "matmultKernel.h"

//#define FOOTPRINT_SIZE BLOCK_SIZE
// Define a gpu kernel to perform matrix multiplication
// of A x B = C.

////// Helper functions start//////  
__device__ float getElement(const Matrix A, int row, int col){
return A.elements[row*A.stride+col];
}
//
__device__ void setElement(Matrix A, int row, int col, float val){
A.elements[row*A.stride*BLOCK_SIZE +col*BLOCK_SIZE] = val;
}

__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}
///// Helper end functions end/////

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
// Block index
int bx = blockIdx.x;
int by = blockIdx.y;

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Index of the first sub-matrix of A processed by the block
int aBegin = wA * BLOCK_SIZE * by;

// Index of the last sub-matrix of A processed by the block
int aEnd   = aBegin + wA - 1;

// Step size used to iterate through the sub-matrices of A
int aStep  = BLOCK_SIZE;

// Index of the first sub-matrix of B processed by the block
int bBegin = BLOCK_SIZE * bx;

// Step size used to iterate through the sub-matrices of B
int bStep  = BLOCK_SIZE * wB;

// Csub is used to store the element of the block sub-matrix
// that is computed by the thread
float Csub[4] = {0, 0, 0, 0}; // Each thread computes 4 elements

// Loop over all the sub-matrices of A and B
// required to compute the block sub-matrix
for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
        Csub[0] += As[ty][k] * Bs[k][tx];
        Csub[1] += As[ty][k] * Bs[k][tx + BLOCK_SIZE];
        Csub[2] += As[ty + BLOCK_SIZE][k] * Bs[k][tx];
        Csub[3] += As[ty + BLOCK_SIZE][k] * Bs[k][tx + BLOCK_SIZE];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
}

// Write the block sub-matrix to device memory;
// each thread writes one element
int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
C[c + wB * ty + tx] = Csub[0];
C[c + wB * ty + tx + BLOCK_SIZE] = Csub[1];
C[c + wB * (ty + BLOCK_SIZE) + tx] = Csub[2];
C[c + wB * (ty + BLOCK_SIZE) + tx + BLOCK_SIZE] = Csub[3];

}