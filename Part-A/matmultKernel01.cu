#include "hip/hip_runtime.h"
/*
For COMS E6998(HPML) Spring 2024
Modified by Yonathan Daniel
*/
#include <stdio.h> // for debugging

#include "matmultKernel.h"

//#define FOOTPRINT_SIZE BLOCK_SIZE
// Define a gpu kernel to perform matrix multiplication
// of A x B = C.

////// Helper functions start//////  
__device__ float getElement(const Matrix A, int row, int col){
return A.elements[row*A.stride+col];
}
//
__device__ void setElement(Matrix A, int row, int col, float val){
A.elements[row*A.stride*BLOCK_SIZE +col*BLOCK_SIZE] = val;
}

__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}
///// Helper end functions end/////
/*
within given block
how to losf into shared mem?

instead of having like a square/rectangular filter
load element from subarray and compute that way

A = | 1 | 2 |
    | 3 | 4 |

To compute C_sub
->
Main difference thread to compute offsets in block
    */
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
// matrix blocks
float *Asub, *Bsub, *Csub;
// Putting these into registers speeds access.
int thread_row = threadIdx.y;
int thread_col = threadIdx.x;

//this is where the thread blocks start in the thread space
int block_row = blockIdx.y;
int block_col = blockIdx.x;

//4 different C values
float Cvalue0 = 0;
float Cvalue1 = 0;
float Cvalue2 = 0;
float Cvalue3 = 0;

/*
re
*/

Csub = &C.elements[C.stride * block_row * FOOTPRINT_SIZE + FOOTPRINT_SIZE * block_col];//&C.elements[C.stride * footprint_row * FOOTPRINT_SIZE + FOOTPRINT_SIZE * footprint_col];

for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){

      Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];//&A.elements[A.stride * FOOTPRINT_SIZE * footprint_row + FOOTPRINT_SIZE * m];
      Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];//&B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * footprint_col];

      __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
      __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];

      shared_A[thread_row][thread_col] = Asub[A.stride * thread_row + thread_col];
      shared_B[thread_row][thread_col] = Bsub[B.stride * thread_row + thread_col]; 

      shared_A[thread_row][thread_col+BLOCK_SIZE] = Asub[A.stride * thread_row + thread_col+BLOCK_SIZE];
      shared_B[thread_row][thread_col+BLOCK_SIZE] = Bsub[B.stride * thread_row + thread_col+BLOCK_SIZE]; 

      shared_A[thread_row+BLOCK_SIZE][thread_col] = Asub[A.stride * (thread_row + BLOCK_SIZE) + thread_col];
      shared_B[thread_row+BLOCK_SIZE][thread_col] = Bsub[B.stride * (thread_row + BLOCK_SIZE) + thread_col]; 

      shared_A[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Asub[A.stride * (thread_row + BLOCK_SIZE) + thread_col+BLOCK_SIZE];
      shared_B[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Bsub[B.stride * (thread_row + BLOCK_SIZE) + thread_col+BLOCK_SIZE]; 
      //make sure all threads 
      __syncthreads();

#pragma unroll
  for( int e = 0; e<FOOTPRINT_SIZE; e++){
//top left and top right
Cvalue0 += shared_A[thread_row][e]*shared_B[e][e];
Cvalue1 += shared_A[thread_row][e]*shared_B[e][thread_col+BLOCK_SIZE];
Cvalue2 += shared_A[thread_row+BLOCK_SIZE][e]*shared_B[e][thread_col];
Cvalue3 += shared_A[thread_row+BLOCK_SIZE][e]*shared_B[e][thread_col+BLOCK_SIZE];

  }

  // Synchronize to ensure all Cvalues have been incremented
  // before reading in the next shared_A AND shared_B BLOCKS
  __syncthreads();




}
// Write Csub to GLOBAL memory.
// Each thread writes its own cell value.

Csub[thread_row * C.stride + thread_col  ] = Cvalue0;
Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE] = Cvalue1;
Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE*C.stride] = Cvalue2;
Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE*C.stride + BLOCK_SIZE] = Cvalue3;
}