#include "hip/hip_runtime.h"
/*
For COMS E6998(HPML) Spring 2024
Modified by Yonathan Daniel
*/
#include <stdio.h> // for debugging

#include "matmultKernel.h"

//#define FOOTPRINT_SIZE BLOCK_SIZE
// Define a gpu kernel to perform matrix multiplication
// of A x B = C.

////// Helper functions start//////  
__device__ float getElement(const Matrix A, int row, int col){
return A.elements[row*A.stride+col];
}
//
__device__ void setElement(Matrix A, int row, int col, float val){
A.elements[row*A.stride*BLOCK_SIZE +col*BLOCK_SIZE] = val;
}

__device__ Matrix getSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}
///// Helper end functions end/////
/*
within given block
how to losf into shared mem?

instead of having like a square/rectangular filter
load element from subarray and compute that way

A = | 1 | 2 |
    | 3 | 4 |

To compute C_sub
->
Main difference thread to compute offsets in block
    */
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];

  // Each thread computes one element of Csub in its copy of CValue
  float Cvalue = 0;

  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){
    // Get Asub and Bsub descriptors

    //printf("A: %d B: %d ", A.stride * BLOCK_SIZE * block_row + FOOTPRINT_SIZE * m, B.stride * FOOTPRINT_SIZE * m + BLOCK_SIZE * block_col);

    Asub = &A.elements[A.stride * BLOCK_SIZE * block_row + FOOTPRINT_SIZE * m];
    Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + BLOCK_SIZE * block_col];

    // Copy ELEMENTS OF  ASub and Bsub into shared memory
    // EACH THREAD loads ONE ELEMENT of ASub and ONE of Bsub
    // Notice: it does not need to be the element it requires to
    //         compute its Cvalue, as long as all elements are 
    //         collaboratively read. 

    // Notice: every thread declares shared_A and shared_B in shared memory
    //         even though a thread block has only one shared_A and one shared_B
    __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
    __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];



{//do this for 4 different indecies // Each thread copies just 4 element of shared_A and one element of shared_B

    for( int r = 0; r<2; r++)
    for( int c = 0; c<2; c++){
    shared_A[thread_row+r*BLOCK_SIZE][thread_col+(BLOCK_SIZE*c)] = Asub[(thread_row+r*BLOCK_SIZE) * A.stride + (thread_col + BLOCK_SIZE*c)];
    shared_B[thread_row+r*BLOCK_SIZE][thread_col+(BLOCK_SIZE*c)] = Bsub[(thread_row+r*BLOCK_SIZE) * B.stride + (thread_col + BLOCK_SIZE*c)];
    }


}

    // Synchronize to ensure all elements are read
    __syncthreads();

    // Do an inproduct of one row of shared_A and one col of shared_B
    // computing one Cvalue by accumulation
#pragma unroll
    for(int e=0; e<FOOTPRINT_SIZE; ++e){


{//this loop reading in of C value needs to be done 4 times for each of the 4 values
       
       for( int r = 0; r<2; r++){ for( int c = 0; c<2; c++){
	       Csub[(thread_row+r*BLOCK_SIZE) * C.stride + (thread_col + BLOCK_SIZE*c)] += shared_A[(thread_row+r*BLOCK_SIZE)][e] * shared_B[e][(thread_col + BLOCK_SIZE*c)];
	       }}
}

       }
    // Synchronize to ensure all Cvalues have been incremented
    // before reading in the next shared_A AND shared_B BLOCKS
    __syncthreads();
  }

  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
  //Csub[thread_row * C.stride + thread_col] = Cvalue;
}